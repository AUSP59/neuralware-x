// SPDX-License-Identifier: Apache-2.0
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>
#include <cstring>

extern "C" {
// Row-major A[m,k] * B[k,n] => C[m,n]
// Returns 0 on success; negative on error.
int nwx_cublas_dgemm(int m, int n, int k, const double* A_host, const double* B_host, double* C_host) {
  hipError_t ce;
  hipblasStatus_t st;
  double *A=NULL, *B=NULL, *C=NULL;
  size_t sA = (size_t)m*k*sizeof(double);
  size_t sB = (size_t)k*n*sizeof(double);
  size_t sC = (size_t)m*n*sizeof(double);

  if ((ce=hipMalloc((void**)&A, sA)) != hipSuccess) return -1;
  if ((ce=hipMalloc((void**)&B, sB)) != hipSuccess) { hipFree(A); return -2; }
  if ((ce=hipMalloc((void**)&C, sC)) != hipSuccess) { hipFree(A); hipFree(B); return -3; }

  if ((ce=hipMemcpy(A, A_host, sA, hipMemcpyHostToDevice)) != hipSuccess) { hipFree(A); hipFree(B); hipFree(C); return -4; }
  if ((ce=hipMemcpy(B, B_host, sB, hipMemcpyHostToDevice)) != hipSuccess) { hipFree(A); hipFree(B); hipFree(C); return -5; }

  hipblasHandle_t h;
  if ((st=hipblasCreate(&h)) != HIPBLAS_STATUS_SUCCESS) { hipFree(A); hipFree(B); hipFree(C); return -6; }

  // Use row-major by swapping operands and dims (C = A*B row-major => C^T = B^T * A^T col-major)
  const double alpha=1.0, beta=0.0;
  st = hipblasDgemm(h, HIPBLAS_OP_T, HIPBLAS_OP_T,
                   n, m, k,
                   &alpha,
                   B, k,  // B^T: (n x k) uses B with leading dim k
                   A, m,  // A^T: (k x m) uses A with leading dim m
                   &beta,
                   C, n); // C^T: (n x m) laid out row-major target with ld=n
  if (st != HIPBLAS_STATUS_SUCCESS) { hipblasDestroy(h); hipFree(A); hipFree(B); hipFree(C); return -7; }

  st = hipblasDestroy(h);
  if ((ce=hipMemcpy(C_host, C, sC, hipMemcpyDeviceToHost)) != hipSuccess) { hipFree(A); hipFree(B); hipFree(C); return -8; }

  hipFree(A); hipFree(B); hipFree(C);
  hipDeviceSynchronize();
  return 0;
}
} // extern "C"
